#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <ATen/ATen.h>

__global__ void custom_operator_my(const float *input,
                                   float *output,
                                   const int height,
                                   const int width)
{
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= width || y >= height)
        return;
    output[y * width + x] = input[y * width + x] * 2;
}

at::Tensor my_custom_operator_mul2(at::Tensor input)
{
    auto output = at::zeros_like(input);
    // one pixel per block
    constexpr int pixel_per_block = 1;
    custom_operator_my<<<dim3(input.size(1) / pixel_per_block, input.size(0) / pixel_per_block), dim3(pixel_per_block, pixel_per_block)>>>(input.data_ptr<float>(), output.data_ptr<float>(), input.size(0), input.size(1));
    return output;
}